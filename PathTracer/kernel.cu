#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
using namespace std;
#include <stdio.h>
#include "cutil_math.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>
#include <unordered_map>

__device__ const double c = 299792458, k = 138064852e-31, PI = 3.141592653589793238463;

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();

	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA error: %s: %s. \n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

// hash function to calculate new seed for each frame
// see http://www.reedbeta.com/blog/2013/01/12/quick-and-easy-gpu-random-numbers-in-d3d11/
uint WangHash(uint a) {
	a = (a ^ 61) ^ (a >> 16);
	a = a + (a << 3);
	a = a ^ (a >> 4);
	a = a * 0x27d4eb2d;
	a = a ^ (a >> 15);
	return a;
}

// "__host__": This function called by CPU and runs on CPU
// "__device__": This function called by GPU and runs on GPU (inside one thread)
// "__global__": This is a kernel function, called by CPU and runs on GPU
// "__constant__": This data won't and can't be modified

// Changing variables
__constant__ float3 cam_right = { 200.0f, 150.0f, 1100.0f };
__constant__ float3 cam_left = { 100.0f, 150.0f, 1100.0f };
#define USING_WAVE 0	// from 0 to 10

// reflection type (DIFFuse, SPECular, REFRactive)
enum Refl_t { DIFF, SPEC, REFR };
// geometry type
enum Geom_t { SPHERE, CONE, TRIANGLE};

// mat name
#define mat_human 0
#define mat_marble 1
#define mat_paint 2
#define mat_glass 3
#define mat_rubber 4
#define mat_brass 5
#define mat_road 6
#define mat_al 7
#define mat_al2o3 8
#define mat_brick 9

__constant__ float wave[11] = {
	7.8576538e+02,
	8.1770000e+02,
	8.6250000e+02,
	9.1025000e+02,
	9.4255000e+02,
	9.7750000e+02,
	1.0277500e+03,
	1.0780000e+03,
	1.1255000e+03,
	1.1860000e+03,
	1.2766667e+03
};


// emiLib[waveNum][matName]
__constant__ float emiLib[11][10] = {
	/*
	human,			marble,			paint,			glass,			rubber,			brass,			road,			al,				al2o3,			brick*/
	9.9000000e-01,	9.5834758e-01,	8.7470001e-01,	5.0455443e-01,	9.2789246e-01,	1.2250251e-01,	9.6426578e-01,	5.5701898e-01,	4.1617280e-02,	9.7773773e-01,
	9.9000000e-01,	9.5462609e-01,	8.8365367e-01,	2.8523451e-01,	9.2827028e-01,	1.1789014e-01,	9.7194589e-01,	5.4616836e-01,	4.1602933e-02,	9.7348785e-01,
	9.9000000e-01,	9.5099592e-01,	9.6279529e-01,	3.8887318e-01,	9.2640468e-01,	1.2078545e-01,	9.6430868e-01,	5.2990503e-01,	4.0821044e-02,	9.6252597e-01,
	9.9000000e-01,	9.5741246e-01,	8.6909910e-01,	4.2252257e-01,	9.2027605e-01,	1.2892990e-01,	9.4494491e-01,	5.1621436e-01,	4.8036999e-02,	9.4693874e-01,
	9.9000000e-01,	9.6385735e-01,	8.5889954e-01,	4.4505789e-01,	9.2317386e-01,	1.3452107e-01,	9.5513005e-01,	5.0484414e-01,	1.4619579e-01,	9.3275042e-01,
	9.9000000e-01,	9.6087765e-01,	9.3344199e-01,	4.7704424e-01,	8.9968776e-01,	1.4311263e-01,	9.5631467e-01,	4.9568769e-01,	2.6974721e-01,	9.1201603e-01,
	9.9000000e-01,	9.5962251e-01,	9.4205163e-01,	5.6399482e-01,	8.6774658e-01,	1.4932587e-01,	9.5258259e-01,	4.7984848e-01,	4.2480553e-01,	8.7901868e-01,
	9.9000000e-01,	9.5305901e-01,	9.4627694e-01,	3.2859562e-01,	8.8061124e-01,	1.4229701e-01,	9.1783893e-01,	4.6578646e-01,	4.7823023e-01,	8.5128884e-01,
	9.9000000e-01,	9.5385122e-01,	9.5199753e-01,	4.2369253e-02,	8.9911606e-01,	1.3455656e-01,	9.1771733e-01,	4.5454008e-01,	5.1389488e-01,	9.0261137e-01,
	9.9000000e-01,	9.5852822e-01,	9.5649050e-01,	2.7487807e-02,	9.1817783e-01,	1.2604779e-01,	9.1884949e-01,	4.3838823e-01,	5.4462383e-01,	9.3754130e-01,
	9.9000000e-01,	9.5240096e-01,	9.5069231e-01,	8.9005827e-02,	9.3104627e-01,	1.1098321e-01,	9.5362853e-01,	4.1783501e-01,	5.6727138e-01,	9.7270040e-01
};

__device__ float BBp (float T, float v)
{
	double h = 2 * PI * 105457180e-42;
	//// 2e8*2*pi*h_bar*c^2
	//float c1 = 1.1910429524674593e-08;
	//// 100*2*pi*h_bar*c/k
	//float c2 = 1.4387773536379256;
	//// BBp = c1*pow(v,3)/(exp(c2*v/T)-1)
	//return float(c1 * pow(v, 3) / (exp(c2 * v / T) - 1.0f));

	return 2e8 * (h * c * c * v * v * v) / (exp(100 * h * c * v / k / T) - 1);
}

struct Ray {
	float3 origin;
	float3 direction;
	// create a ray
	__device__ Ray(float3 o_, float3 d_) : origin(o_), direction(d_) {}
};

struct Hit
{
	float hitDist;		//hitDistance
	float3 normal;
	float3 oriNormal;	// oriented normal (for rafraction)
	float3 nextDir;		// direction for next segment
	Refl_t reflectType;
	Geom_t geomtryType;
	int geomID;
	float3 color;
	float3 emission;
	__device__ void Init() {
		hitDist = 1e20;
		normal = make_float3(0.0f);
		oriNormal = make_float3(0.0f);
		nextDir = make_float3(0.0f);
		reflectType = DIFF;
		geomtryType = SPHERE;
		geomID = -1;
		color = make_float3(0.0f);
		emission = make_float3(0.0f);
	}
};

struct Sphere {

	float radius;
	float3 position;
	int matName;
	float temperature;
	Refl_t reflectType;	//DIFF, SPEC, REFR
	__device__ float intersect(const Ray& ray) const { // returns distance, 0 if nohit 

		// Ray/sphere intersection
		// Quadratic formula required to solve ax^2 + bx + c = 0 
		// Solution x = (-b +- sqrt(b*b - 4ac)) / 2a
		// Solve t^2*d.d + 2*t*(o-p).d + (o-p).(o-p)-R^2 = 0 

		float3 op = position - ray.origin;
		float t, epsilon = 0.01f;
		float b = dot(op, ray.direction);
		float disc = b * b - dot(op, op) + radius * radius; // discriminant
		if (disc < 0) return 0; else disc = sqrtf(disc);
		return (t = b - disc) > epsilon ? t : ((t = b + disc) > epsilon ? t : 0);
	}
};

struct Cone {
	float theta;
	float3 tip, axis;
	int matName;
	float temperature;
	Refl_t reflectType;	//DIFF, SPEC, REFR
	__device__ float intersect(const Ray& ray) const { // returns distance, 0 if nohit  

		float3 co = ray.origin - tip; float cos2t = cos(theta*PI/180.0f); cos2t *= cos2t;
		float t, dotDV = dot(ray.direction, axis), dotCOV = dot(co, axis);
		float a = dotDV * dotDV - cos2t, b = 2.0f * (dotDV * dotCOV - dot(ray.direction, co) * cos2t),
			c = dotCOV * dotCOV - dot(co, co) * cos2t, delta = b * b - 4 * a * c;
		if (delta <= 0.0f) return 0; else delta = sqrt(delta);
		t = (-b + delta) / 2.0f / a > 0.01f ? (-b + delta) / 2.0f / a : max((-b - delta) / 2.0f / a, 0.0f);
		float3 hit = ray.origin + t * ray.direction;
		if (dot(hit - tip, axis) <= 0.0f) return 0;
		return t;
	}
};

__device__ float TriangleIntersect(const Ray& ray, float3 vert0, float3 vert1, float3 vert2, float &u, float &v)
{
	// find vectors for two edges sharing vert0
	float3 edge1 = vert1 - vert0;
	float3 edge2 = vert2 - vert0;
	float t;
	// begin calculating determinant - also used to calculate U parameter
	float3 pvec = cross(ray.direction, edge2);
	// if determinant is near zero, ray lies in plane of triangle
	float det = dot(edge1, pvec);
	// use backface culling
	if (det < 0.01f)
		return 0;
	float inv_det = 1.0f / det;
	// calculate distance from vert0 to ray origin
	float3 tvec = ray.origin - vert0;
	// calculate U parameter and test bounds
	u = dot(tvec, pvec) * inv_det;
	if (u < 0.0 || u > 1.0f)
		return 0;
	// prepare to test V parameter
	float3 qvec = cross(tvec, edge1);
	// calculate V parameter and test bounds
	v = dot(ray.direction, qvec) * inv_det;
	if (v < 0.0 || u + v > 1.0f)
		return 0;
	// calculate t, ray intersects triangle
	t = dot(edge2, qvec) * inv_det;
	return t;
}

#define room_width 300.0f
#define room_height 300.0f
#define room_depth 1200.0f
__constant__ Sphere spheres[] = {
	/* cornell box
	{radius	position												matName		temperature			reflectType*/
	{1e5f,	{-1e5f, 0.0f, 0.0f},									mat_brick,	20.0f + 273.15f,	DIFF},// left wall
	{1e5f,	{1e5f + room_width, 0.0f, 0.0f},						mat_brick,	20.0f + 273.15f,	DIFF},// right wall
	{1e5f,	{0.0f, 0.0f, -1e5f},									mat_brick,	20.0f + 273.15f,	DIFF},// back wall
	{1e5f,	{0.0f, 0.0f, 1e5f + room_depth},						mat_brick,	20.0f + 273.15f,	DIFF},// front wall
	{1e5f,	{0.0f, -1e5f, 0.0f},									mat_road,	20.0f + 273.15f,	DIFF},// floor
	{1e5f,	{0.0f, 1e5f + room_height, 0.0f},						mat_brick,	20.0f + 273.15f,	DIFF},// ceiling  
	{50.0f,	{200.0f ,50.0f, 700.0f},								mat_al,		72.5f + 273.15f,	DIFF},// sphere 
	{600.0f,{room_width/2 ,room_height+600.0f-2.0f, room_depth/2},	mat_glass,	100.0f + 273.15f,	DIFF} // lamp 
};

__constant__ Cone cones[] = {
	/*
	theta	tip							axis					matName		temperature			reflectType*/
	{15,	{100.0f, 80.0f, 500.0f},	{0.0f, -1.0f, 0.0f},	mat_rubber,	37.0f + 273.15f,	DIFF}
};

__device__ inline bool intersect_scene(const Ray& ray, Hit& bestHit, 
	int vertsNum, float3* scene_verts, int objsNum, int* scene_objs_info,
	float2* scene_uvs, float3* scene_normals,
	int texNum, int* tex_wh, float3* tex_data, float* emiList)
{
	float d = 1e20;
	float INF = 1e20;

	//// intersect all spheres in the scene
	//float spheresNum = sizeof(spheres) / sizeof(Sphere);
	//for (int i = 0; i < spheresNum; i++)  // for all spheres in scene
	//{
	//	// keep track of distance from origin to closest intersection point
	//	if ((d = spheres[i].intersect(ray)) && d < bestHit.hitDist && d > 0)
	//	{ 
	//		bestHit.hitDist = d;
	//		bestHit.geomtryType = SPHERE;
	//		bestHit.geomID = i;
	//	}
	//}

	//// intersect all cones in the scene
	//float conesNum = sizeof(cones) / sizeof(Cone);
	//for (int i = 0; i < conesNum; i++)  // for all cones in scene
	//{
	//	// keep track of distance from origin to closest intersection point
	//	if ((d = cones[i].intersect(ray)) && d < bestHit.hitDist && d > 0)
	//	{
	//		bestHit.hitDist = d;
	//		bestHit.geomtryType = CONE;
	//		bestHit.geomID = i;
	//	}
	//}

	// intersect all triangles in the scene
	int currentObj = 0;	// current object max vert = scene_objs_info[currentObj]
	int facesNum = vertsNum / 3;
	for (int i = 0; i < facesNum; i++)
	{
		float3 v0 = scene_verts[i * 3];
		float3 v1 = scene_verts[i * 3 + 1];
		float3 v2 = scene_verts[i * 3 + 2];
		int currentVert = i * 3 + 2;
		// u, v, 1-u-v; 
		float u = 0; 
		float v = 0;
		// which object?
		if (currentObj + 1 < objsNum)
		{
			// move to next obj
			if (currentVert >= scene_objs_info[(currentObj + 1) * 6]) currentObj++;
		}
		if ((d = TriangleIntersect(ray, v0, v1, v2, u, v)) && d < bestHit.hitDist && d > 0)
		{
			float2 uv0 = scene_uvs[i * 3];
			float2 uv1 = scene_uvs[i * 3 + 1];
			float2 uv2 = scene_uvs[i * 3 + 2];
			float w = 1 - u - v;
			float2 uv = w*uv0 + u*uv1 + v*uv2;
			//bestHit.color = make_float3(uv.x, uv.y, 0);
			bestHit.reflectType = DIFF;
			if (scene_objs_info[currentObj * 6 + 4] == 1) bestHit.reflectType = SPEC;
			else if (scene_objs_info[currentObj * 6 + 4] == 2) bestHit.reflectType = REFR;
			// do not have a normal texture
			if (scene_objs_info[currentObj * 6 + 2] == -1)
			{
				bestHit.normal = normalize(scene_normals[i]);
			}
			else
			{
				// find normal tex in all textures
				int texIndex = scene_objs_info[currentObj * 6 + 2];
				int texWidth = tex_wh[texIndex * 2];
				int texHeight = tex_wh[texIndex * 2 + 1];
				int offset = 0;	// get pixel offset in tex_data
				for (int t = 0; t < texIndex; t++)
				{
					offset += tex_wh[t * 2] * tex_wh[t * 2 + 1];
				}
				// map current uv(float2) to index in tex_data[]
				int u_index = uv.x * texWidth;
				int v_index = uv.y * texHeight;
				// map the color in tex_data[offset + u_index*texWidth + v_index] to normal
				bestHit.normal = normalize(tex_data[offset + v_index * texWidth + u_index] * 2.0f - 1.0f);
			}
			// which emi source?
			// 0: mat, 1: tex, 2: value
			if (scene_objs_info[currentObj * 6 + 5] == 0)
			{
				bestHit.color = make_float3(1.0f);
			}
			else if (scene_objs_info[currentObj * 6 + 5] == 1 && scene_objs_info[currentObj * 6 + 3] != -1)
			{
				// find tex in all textures
				int texIndex = scene_objs_info[currentObj * 6 + 3];
				int texWidth = tex_wh[texIndex * 2];
				int texHeight = tex_wh[texIndex * 2 + 1];
				int offset = 0;	// get pixel offset in tex_data
				for (int t = 0; t < texIndex; t++)
				{
					offset += tex_wh[t * 2] * tex_wh[t * 2 + 1];
				}
				// map current uv(float2) to index in tex_data[]
				int u_index = uv.x * texWidth;
				int v_index = uv.y * texHeight;
				// map the color in tex_data[offset + v_index * texWidth + u_index] to emissivity
				bestHit.color = tex_data[offset + v_index * texWidth + u_index];
			}
			
			bestHit.hitDist = d;
			bestHit.geomtryType = TRIANGLE;
			bestHit.oriNormal = dot(bestHit.normal, ray.direction) < 0.0f ? bestHit.normal : bestHit.normal * -1.0f;
		}
	}
	

	// t is distance to closest intersection of ray with all primitives in the scene
	if (bestHit.hitDist < INF)
	{
		float3 hitPostion = ray.origin + ray.direction * bestHit.hitDist;
		switch (bestHit.geomtryType)
		{
		case SPHERE:
			bestHit.normal = normalize(hitPostion - spheres[bestHit.geomID].position);
			bestHit.oriNormal = dot(bestHit.normal, ray.direction) < 0.0f ? bestHit.normal : bestHit.normal * -1.0f;
			bestHit.reflectType = spheres[bestHit.geomID].reflectType;
			break;
		case CONE:
			float3 cp = hitPostion - cones[bestHit.geomID].tip;
			bestHit.normal = normalize(cp * dot(cones[bestHit.geomID].axis, cp) / dot(cp, cp) - cones[bestHit.geomID].axis);
			bestHit.oriNormal = dot(bestHit.normal, ray.direction) < 0.0f ? bestHit.normal : bestHit.normal * -1.0f;
			bestHit.reflectType = cones[bestHit.geomID].reflectType;
			break;
		case TRIANGLE:
			
			break;
		default:
			break;
		}
		return true;
	}
	else return false;
}

// result of radiance:
// e0+rt0*(e1+rt1*(e2+rt2*(e3...)))
// save e0 and rt0 in this array
struct RecursionData
{
	float emission;
	float reflectivity;
	__device__ void add(float emi, float rt)
	{
		emission = emi;
		reflectivity = rt;
	}
	__device__ void init()
	{
		emission = 0.0f;
		reflectivity = 0.0f;
	}
};

// radiance function
// compute path bounces in scene and accumulate returned color from each path sgment
__device__ float3 radiance(Ray& ray, hiprandState* randstate, int frameNum, int waveNum, int index, 
	int vertsNum, float3* scene_verts, int objsNum, int* scene_objs_info,
	float2* scene_uvs, float3* scene_normals,
	int texNum, int* tex_wh, float3* tex_data,
	int type, float* emiList) {

	Hit bestHit;
	// accumulated color for current pixel
	float3 colorMask = make_float3(1.0f);
	// accumulated color for current pixel
	float3 accuIntensity = make_float3(0.0f);

	//// hit debug
	//bestHit.Init();
	//if (!intersect_scene(ray, bestHit, vertsNum, scene_verts, objsNum, scene_objs_info, scene_uvs, scene_normals,
	//	texNum, tex_wh, tex_data, emiList))
	//	return make_float3(0.0f); // if miss, return black
	//else
	//{
	//	return bestHit.color;
	//}
	//// hit debug end

	int bounces = 0;
	while(bounces < 5 || hiprand_uniform(randstate) < 0.5f)
	{  
		if (bounces >= 10) break;
		bounces++;
		bestHit.Init();
		float emi = 0.0f;
		float rt = 0.0f;
		// intersect ray with scene
		if (!intersect_scene(ray, bestHit, vertsNum, scene_verts, objsNum, scene_objs_info, scene_uvs, scene_normals,
							texNum, tex_wh, tex_data, emiList))
		{
			// sky color
			bestHit.color = make_float3(0.0f);
			bestHit.emission = make_float3(0.8f);
			accuIntensity += colorMask * bestHit.emission;
			break; // if miss STOP looping, will influnce the output of recuData since already return 
		}

		// else: we've got a hit with a scene primitive
		accuIntensity += colorMask * bestHit.emission;

		float3 hitPosition = ray.origin + ray.direction * bestHit.hitDist;

		// ideal diffuse reflection
		if (bestHit.reflectType == DIFF)
		{
			// create 2 random numbers
			float r1 = 2 * PI * hiprand_uniform(randstate);
			float r2 = hiprand_uniform(randstate);
			float r2s = sqrtf(r2);

			// compute orthonormal coordinate frame uvw with hitpoint as origin 
			float3 w = bestHit.oriNormal;
			float3 u = normalize(cross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
			float3 v = cross(w, u);

			// compute cosine weighted random ray direction on hemisphere 
			bestHit.nextDir = normalize(u * cos(r1) * r2s + v * sin(r1) * r2s + w * sqrtf(1 - r2));

			// offset origin next path segment to prevent self intersection
			hitPosition += bestHit.oriNormal * 0.03;

			colorMask *= bestHit.color;
		}
		// ideal specular reflection
		if (bestHit.reflectType == SPEC)
		{

			// reflect
			bestHit.nextDir = ray.direction - 2.0f * bestHit.normal * dot(bestHit.normal, ray.direction);

			// offset origin next path segment to prevent self intersection
			hitPosition += bestHit.oriNormal * 0.01;

			// multiply color to the object
			colorMask *= bestHit.color;
		}

		// ideal refraction (based on smallpt code by Kevin Beason)
		if (bestHit.reflectType == REFR)
		{

			bool into = dot(bestHit.normal, bestHit.oriNormal) > 0; // is ray entering or leaving refractive material?
			float nc = 1.0f;  // Index of Refraction air
			float nt = 1.5f;  // Index of Refraction glass/water
			float nnt = into ? nc / nt : nt / nc;  // IOR ratio of refractive materials
			float ddn = dot(ray.direction, bestHit.oriNormal);
			float cos2t = 1.0f - nnt * nnt * (1.f - ddn * ddn);

			if (cos2t < 0.0f) // total internal reflection 
			{
				bestHit.nextDir = reflect(ray.direction, bestHit.normal); //d = r.dir - 2.0f * n * dot(n, r.dir);
				hitPosition += bestHit.oriNormal * 0.01f;
			}
			else // cos2t > 0
			{
				// compute direction of transmission ray
				float3 tdir = normalize(ray.direction * nnt - bestHit.normal * ((into ? 1 : -1) * (ddn * nnt + sqrtf(cos2t))));

				float R0 = (nt - nc) * (nt - nc) / (nt + nc) * (nt + nc);
				float c = 1.f - (into ? -ddn : dot(tdir, bestHit.normal));
				float Re = R0 + (1.f - R0) * c * c * c * c * c;
				float Tr = 1 - Re; // Transmission
				float P = .25f + .5f * Re;
				float RP = Re / P;
				float TP = Tr / (1.f - P);

				// randomly choose reflection or transmission ray
				if (hiprand_uniform(randstate) < 0.25) // reflection ray
				{
					colorMask *= RP;
					bestHit.nextDir = reflect(ray.direction, bestHit.normal);
					hitPosition += bestHit.oriNormal * 0.01f;
				}
				else // transmission ray
				{
					colorMask *= TP;
					bestHit.nextDir = tdir; //r = Ray(x, tdir); 
					hitPosition += bestHit.oriNormal * 0.0005f; // epsilon must be small to avoid artefacts
				}
			}
		}
		// set up origin and direction of next path segment
		ray.origin = hitPosition;
		ray.direction = bestHit.nextDir;
	}

	return accuIntensity;
}

__device__ float3 gammaCorrect(float3 c)
{
	float3 g;
	g.x = pow(c.x, 1 / 2.2f);
	g.y = pow(c.y, 1 / 2.2f);
	g.z = pow(c.z, 1 / 2.2f);
	return g;
}

__global__ void render(float3 *result, float3* accumbuffer, hiprandState* randSt, 
	int width, int height, int frameNum, int HashedFrameNum, bool camAtRight, int waveNum, 
	int vertsNum, float3* scene_verts, int objsNum, int* scene_objs_info,
	float2* scene_uvs, float3* scene_normals,
	int texNum, int* tex_wh, float3* tex_data,
	int type, float* emiList)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= width) || (j >= height)) 
		return;
	// unique id for the pixel
	int index = j * width + i;
	if (frameNum == 0)	//init
	{
		accumbuffer[index] = make_float3(0.0);
	}
	else
	{
		// create random number generator, see RichieSams blogspot
		hiprandState randState; // state of the random number generator, to prevent repetition, need refresh per frame
		hiprand_init(HashedFrameNum + index, 0, 0, &randState);
		float3 pixelColor = make_float3(0);
		// offset inside each pixel
		float offsetX = hiprand_uniform(&randState);	// get random float between (0, 1)
		float offsetY = hiprand_uniform(&randState);
		// uv(-0.5, 0.5)
		float2 uv = make_float2((i + offsetX) / width, (j + offsetY) / height) - make_float2(0.5f, 0.5f);
		float3 camPos;
		if (camAtRight) camPos = cam_right;
		else camPos = cam_left;
		Ray cam(camPos, normalize(make_float3(0.0f, 0.0f, -1.0f)));
		float3 screen = make_float3(uv.x * width + room_width / 2.0f, -uv.y * height + room_width / 2.0f, 1100.0f - (width / 2.0f) * 1.73205080757f);
		// screen x offset
		if (camAtRight)
		{
			screen += make_float3(50.0f, 0.0f, 0.0f);
		} 
		else
		{
			screen -= make_float3(50.0f, 0.0f, 0.0f);
		}
		float3 dir = normalize(screen - cam.origin);
		//result[index] = make_float3(dir.x);
		float3 intensity = radiance(Ray(cam.origin, dir), &randState, frameNum, waveNum, index, 
			vertsNum, scene_verts, objsNum, scene_objs_info, scene_uvs, scene_normals,
			texNum, tex_wh, tex_data,
			type, emiList);
		pixelColor = intensity;

		accumbuffer[index] += pixelColor;
	}
	float3 tempCol = accumbuffer[index] / (float)frameNum;
	//tempCol = gammaCorrect(tempCol);

	result[index] = tempCol;
}

extern "C" void launch_kernel(float3* result, float3* accumbuffer, hiprandState* randState, 
	unsigned int w, unsigned int h, unsigned int frame, 
	bool camAtRight, int waveNum, 
	int vertsNum, float3* scene_verts, 
	int objsNum, int* scene_objs_info,
	float2* scene_uvs, float3* scene_normals,
	int texNum, int* tex_wh, float3* tex_data,
	int type, float* emiList) {

	//set thread number
	int tx = 16;
	int ty = 16;

	dim3 blocks(w / tx + 1, h / ty + 1);
	dim3 threads(tx, ty);
	
	render <<<blocks, threads >>> (result, accumbuffer, randState, w, h, frame, WangHash(frame), 
		camAtRight, waveNum,
		vertsNum, scene_verts, objsNum, scene_objs_info, scene_uvs, scene_normals,
		texNum, tex_wh, tex_data,
		type, emiList);

	hipDeviceSynchronize();
	checkCUDAError("kernel failed!");
}

